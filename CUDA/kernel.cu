/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 128

__global__ void naiveMM(int m, int n, int k, const float *A, const float *B, float* C){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    if(id<m*n) 
    {
        int row = (int)(id / m);
        int col = id % n;
        //printf("row %d col %d\n",row,col);
        for(int i = 0; i < k; i++) 
        {
            //printf("row %d col %d k %d i %d n %d A %d B %d %f %f\n",row,col,k,i,n,row*k+i,i*n+col, A[row*k+i],B[i*n+col]);
            sum += A[row * k + i] * B[i * n + col];
            //printf("sum %f\n",sum);
        }
        //printf("%d %d\n",id,sum);
        C[id] = sum;
    }
}

/*
__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    __shared__ float a[TILE_SIZE][TILE_SIZE];
    __shared__ float b[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_SIZE + ty,
       Col = bx * TILE_SIZE + tx;
    float Pvalue = 0;

    for (int i = 0; i < (k-1)/TILE_SIZE+1; ++i) {
        a[ty][tx] = A[Row*k + i*TILE_SIZE+tx];
        b[ty][tx] = B[(i*TILE_SIZE+ty)*n+Col];

       __syncthreads();
       for (int j = 0; j < TILE_SIZE; ++j)
          Pvalue += a[ty][j] * b[j][tx];
       __syncthreads();
    }
    if (Row < m && Col < n)
       C[Row*n+Col] = Pvalue;
}
*/

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE
    int gridSize = (int)ceil((float) (m*n)/BLOCK_SIZE);;

    dim3 dimGrid((n-1)/TILE_SIZE+1, (m-1)/TILE_SIZE+1, 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);



    // Invoke CUDA kernel -----------------------------------------------------
    //<<<num of blocks, num of threads per block

    //INSERT CODE HERE
    //printf("dimGrid1 %d dimGrid2 %d blocksize %d\n",(n-1)/TILE_SIZE+1, (m-1)/TILE_SIZE+1, TILE_SIZE);
    //mysgemm<<<dimGrid,dimBlock>>>(m,n,k,A,B,C);
    //if(which==1){
        //printf("gridSize %d blocksize %d\n",gridSize,BLOCK_SIZE);
    naiveMM<<<gridSize,BLOCK_SIZE>>>(m,n,k,A,B,C);
    //}

}


